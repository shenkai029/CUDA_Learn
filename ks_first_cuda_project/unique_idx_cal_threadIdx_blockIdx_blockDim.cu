#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void unique_idx_calc_threadIdx(int* input)
{
	int tid = threadIdx.x;
	printf("threadIdx : %d, value : %d \n", tid, input[tid]);
}

__global__ void unique_gid_calculation(int* input)
{
	int tid = threadIdx.x;
	int offset = blockIdx.x * blockDim.x;
	int gid = tid + offset;
	printf("blockIdx.x : %d, threadIdx : %d, gid: %d, value : %d \n", 
		blockIdx.x, tid, gid, input[gid]);
}

__global__ void unique_gid_calculation_2d(int* input)
{
	int tid = threadIdx.x;
	int block_offset = blockIdx.x * blockDim.x;
	int row_offset = blockDim.x * gridDim.x * blockIdx.y;
	int gid = tid + block_offset + row_offset;
	printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx : %d, gid: %d, value : %d \n",
		blockIdx.x, blockIdx.y, tid, gid, input[gid]);
}

__global__ void unique_gid_calculation_2d_2d(int* input)
{
	int tid = blockDim.x * threadIdx.y + threadIdx.x;

	int num_thread_in_block = blockDim.x * blockDim.y;
	int block_offset = num_thread_in_block * blockIdx.x;

	int num_thread_in_row = num_thread_in_block * gridDim.x;
	int row_offset = num_thread_in_row * blockIdx.y;

	int gid = tid + block_offset + row_offset;
	printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx : %d, gid: %d, value : %d \n",
		blockIdx.x, blockIdx.y, tid, gid, input[gid]);
}


//int main()
//{
//	int array_size = 16;
//	int array_byte_size = sizeof(int) * array_size;
//	int h_data[] = { 23, 9, 4, 53, 65, 12, 1, 33, 87, 45, 23, 13, 342, 56, 44, 99 };
//
//	for (int i = 0; i < array_size; i++)
//	{
//		printf("%d ", h_data[i]);
//	}
//	printf("\n \n");
//
//	int* d_data;
//	cudaMalloc((void**)&d_data, array_byte_size);
//	cudaMemcpy(d_data, h_data, array_byte_size, cudaMemcpyHostToDevice);
//
//	dim3 block(2, 2);
//	dim3 grid(2, 2);
//
//	unique_gid_calculation_2d_2d << <grid, block >> > (d_data);
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//	return 0;
//}