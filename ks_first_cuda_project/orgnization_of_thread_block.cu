#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void print_details()
{
	printf("threadIdx.x : %d, threadIdx.x : %d, threadIdx.z : %d, blockDim.x : %d, blockDim.y : %d, gridDim.x : %d, gridDmi.y : %d \n",
		threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
}

//int main()
//{
//	int nx, ny;
//	nx = 16;
//	ny = 16;
//	dim3 block(8, 8);
//	dim3 grid(nx / block.x, ny / block.y);
//
//	print_details << <grid, block >> > ();
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//	return 0;
//}