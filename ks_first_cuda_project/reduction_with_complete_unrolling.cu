#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "common.h"
#include "cuda_common.cuh"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void reduction_kernel_complete_unrolling(int* int_array,
	int* temp_array, int size)
{
	int tid = threadIdx.x;

	//element index for this thread
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	//local data pointer
	int* i_data = int_array + blockDim.x * blockIdx.x;

	int active_threads = blockDim.x;

	if (active_threads == 1024 && tid < 512)
		i_data[tid] += i_data[tid + 512], active_threads /= 2;
	__syncthreads();

	if (active_threads == 512 && tid < 256)
		i_data[tid] += i_data[tid + 256], active_threads /= 2;
	__syncthreads();

	if (active_threads == 256 && tid < 128)
		i_data[tid] += i_data[tid + 128], active_threads /= 2;
	__syncthreads();

	if (active_threads == 128 && tid < 64)
		i_data[tid] += i_data[tid + 64], active_threads /= 2;
	__syncthreads();

	if (tid < 32)
	{
		volatile int* vsmem = i_data;
		vsmem[tid] += vsmem[tid + 32];
		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid] += vsmem[tid + 1];
	}

	if (tid == 0)
	{
		temp_array[blockIdx.x] = i_data[0];
	}
}

//int main(int argc, char ** argv)
//{
//	printf("Running parallel reduction with complete unrolling kernel \n");
//
//	int size = 1 << 22;
//	int byte_size = size * sizeof(int);
//	int block_size = 1024;
//
//	int * h_input, *h_ref;
//	h_input = (int*)malloc(byte_size);
//
//	initialize(h_input, size, INIT_RANDOM);
//
//	int cpu_result = reduction_cpu(h_input, size);
//
//	dim3 block(block_size);
//	dim3 grid(size / block_size);
//
//	printf("Kernel launch parameters || grid : %d, block : %d \n", grid.x, block.x);
//
//	int temp_array_byte_size = sizeof(int)* grid.x;
//
//	h_ref = (int*)malloc(temp_array_byte_size);
//
//	int * d_input, *d_temp;
//	gpuErrChk(hipMalloc((void**)&d_input, byte_size));
//	gpuErrChk(hipMalloc((void**)&d_temp, temp_array_byte_size));
//
//	gpuErrChk(hipMemset(d_temp, 0, temp_array_byte_size));
//	gpuErrChk(hipMemcpy(d_input, h_input, byte_size,
//		hipMemcpyHostToDevice));
//
//	reduction_kernel_complete_unrolling <<< grid, block >> > (d_input, d_temp, size);
//
//	gpuErrChk(hipDeviceSynchronize());
//	gpuErrChk(hipMemcpy(h_ref, d_temp, temp_array_byte_size, hipMemcpyDeviceToHost));
//
//	int gpu_result = 0;
//	for (int i = 0; i < grid.x; i++)
//	{
//		gpu_result += h_ref[i];
//	}
//
//	compare_results(gpu_result, cpu_result);
//
//	gpuErrChk(hipFree(d_input));
//	gpuErrChk(hipFree(d_temp));
//	free(h_input);
//	free(h_ref);
//
//	gpuErrChk(hipDeviceReset());
//	return 0;
//}