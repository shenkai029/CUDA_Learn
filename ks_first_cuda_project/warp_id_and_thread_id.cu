#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void print_detial_of_warps()
{
	int gid = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;

	int warp_id = threadIdx.x / 32;

	int gbid = blockIdx.y * gridDim.x + blockIdx.x;

	printf("tid : %d, bid.x : %d, bid.y : %d, gid : %d, warp_id : %d, gbid : %d",
		threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, gbid);
}

//int main()
//{
//	dim3 block(42);
//	dim3 grid(2, 2);
//
//	print_detial_of_warps << <grid, block >> > ();
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//	return EXIT_SUCCESS;
//}