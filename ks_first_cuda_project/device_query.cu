#include "hip/hip_runtime.h"


#include "stdio.h"

void query_device()
{
	int deviceCnt = 0;
	hipGetDeviceCount(&deviceCnt);

	if (deviceCnt == 0)
	{
		printf("No CUDA support device found");
	}

	int devNo = 0;
	hipDeviceProp_t iProp;
	hipGetDeviceProperties(&iProp, devNo);

	printf(" Device %d:			%s\n", devNo, iProp.name);
	printf(" Number of multiprocessors:								%d\n", iProp.multiProcessorCount);
	printf(" Clock rate:											%d\n", iProp.multiProcessorCount);
	printf(" Compute capability:									%d.%d\n", iProp.major, iProp.minor);
	printf(" Total amount of global memory:							%4.2f KB\n", iProp.totalGlobalMem / 1024.0);
	printf(" Total amount of constant memory:						%4.2f KB\n", iProp.totalConstMem / 1024.0);
	printf(" Total amount of shared memeory per block:				%4.2f KB\n", iProp.sharedMemPerBlock / 1024.0);
	printf(" Total amount of shared memeory per MP:					%4.2f KB\n", iProp.sharedMemPerMultiprocessor / 1024.0);
	printf(" Total number of registers available per block:			%d\n", iProp.regsPerBlock);
	printf(" Warp size:												%d\n", iProp.warpSize);
	printf(" Maximum number of thread per block:					%d\n", iProp.maxThreadsPerBlock);
	printf(" Maximum number of thread per MP:						%d\n", iProp.maxThreadsPerMultiProcessor);
	printf(" Maximum Grid size:									(%d, %d, %d)\n", iProp.maxGridSize[0], iProp.maxGridSize[1], iProp.maxGridSize[2]);
	printf(" Maximum Block dimension:								(%d, %d, %d)\n", iProp.maxThreadsDim[0], iProp.maxThreadsDim[1], iProp.maxThreadsDim[2]);
}

int main()
{
	query_device();
	return 0;
}