#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "common.h"
#include "cuda_common.cuh"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void reduction_kernel_warp_unrolling(int* int_array,
	int* temp_array, int size)
{
	int tid = threadIdx.x;

	//element index for this thread
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	//local data pointer
	int* i_data = int_array + blockDim.x * blockIdx.x;

	for (int offset = blockDim.x / 2; offset > 64; offset = offset / 2)
	{
		if (tid < offset)
		{
			i_data[tid] += i_data[tid + offset];
		}
		__syncthreads();
	}

	if (tid < 32)
	{
		// When multiple threads are accessing a shared variable, and you want to ensure that the most 
		// up-to-date value is always used, you can use volatile variables
		// in this example, we use volatile here to make sure after all thread execute lines to sum value
		// from the offset location, the array should have most up-to-date value to perform next sum for all thread
		volatile int* vsmem = i_data;
		vsmem[tid] += vsmem[tid + 32];
		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid] += vsmem[tid + 1];
	}

	if (tid == 0)
	{
		temp_array[blockIdx.x] = i_data[0];
	}
}


//int main(int argc, char ** argv)
//{
//	printf("Running parallel reduction with warp unrolling kernel \n");
//
//	int size = 1 << 22;
//	int byte_size = size * sizeof(int);
//	int block_size = 512;
//
//	int * h_input, *h_ref;
//	h_input = (int*)malloc(byte_size);
//
//	initialize(h_input, size, INIT_RANDOM);
//
//	int cpu_result = reduction_cpu(h_input, size);
//
//	dim3 block(block_size);
//	dim3 grid(size / block_size);
//
//	printf("Kernel launch parameters || grid : %d, block : %d \n", grid.x, block.x);
//
//	int temp_array_byte_size = sizeof(int)* grid.x;
//
//	h_ref = (int*)malloc(temp_array_byte_size);
//
//	int * d_input, *d_temp;
//	gpuErrChk(hipMalloc((void**)&d_input, byte_size));
//	gpuErrChk(hipMalloc((void**)&d_temp, temp_array_byte_size));
//
//	gpuErrChk(hipMemset(d_temp, 0, temp_array_byte_size));
//	gpuErrChk(hipMemcpy(d_input, h_input, byte_size,
//		hipMemcpyHostToDevice));
//
//	reduction_kernel_warp_unrolling <<< grid, block >> > (d_input, d_temp, size);
//
//	gpuErrChk(hipDeviceSynchronize());
//	gpuErrChk(hipMemcpy(h_ref, d_temp, temp_array_byte_size, hipMemcpyDeviceToHost));
//
//	int gpu_result = 0;
//	for (int i = 0; i < grid.x; i++)
//	{
//		gpu_result += h_ref[i];
//	}
//
//	compare_results(gpu_result, cpu_result);
//
//	gpuErrChk(hipFree(d_input));
//	gpuErrChk(hipFree(d_temp));
//	free(h_input);
//	free(h_ref);
//
//	gpuErrChk(hipDeviceReset());
//	return 0;
//}