#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void print_details_3d()
{
	printf("threadIdx.x : %d, threadIdx.x : %d, threadIdx.z : %d, blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, blockDim.x : %d, blockDim.y : %d, gridDim.x : %d, gridDmi.y : %d \n",
		threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
}

//int main()
//{
//	int nx, ny, nz;
//	nx = 8;
//	ny = 8;
//	nz = 8;
//	dim3 block(4, 4, 4);
//	dim3 grid(nx / block.x, ny / block.y, nz / block.z);
//
//	print_details_3d << <grid, block >> > ();
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//	return 0;
//}