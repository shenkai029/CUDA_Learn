#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

//__global__ void unique_gid_calculation_2d_2d(int* input)
//{
//	int tid = blockDim.x * threadIdx.y + threadIdx.x;
//
//	int num_thread_in_block = blockDim.x * blockDim.y;
//	int block_offset = num_thread_in_block * blockIdx.x;
//
//	int num_thread_in_row = num_thread_in_block * gridDim.x;
//	int row_offset = num_thread_in_row * blockIdx.y;
//
//	int gid = tid + block_offset + row_offset;
//	printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx : %d, gid: %d, value : %d \n",
//		blockIdx.x, blockIdx.y, tid, gid, input[gid]);
//}

__global__ void unique_gid_calculation_3d(int* input)
{
	int tid = blockDim.x * blockDim.y * threadIdx.z + blockDim.x * threadIdx.y + threadIdx.x;

	int num_thread_in_block = blockDim.x * blockDim.y * blockDim.z;
	int block_offset = num_thread_in_block * blockIdx.x;

	int num_thread_in_row = num_thread_in_block * gridDim.x;
	int row_offset = num_thread_in_row * blockIdx.y;

	int num_thread_in_plane = num_thread_in_row * gridDim.y;
	int plane_offset = num_thread_in_plane * blockIdx.z;

	int gid = tid + block_offset + row_offset + plane_offset;
	printf("blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d,threadIdx : %d, gid: %d, value : %d \n",
		blockIdx.x, blockIdx.y, blockIdx.z, tid, gid, input[gid]);
}


//int main()
//{
//	const int array_size = 64;
//	int array_byte_size = sizeof(int) * array_size;
//	int h_data[array_size];
//
//	time_t t;
//	srand((unsigned)time(&t));
//	for (int i = 0; i < array_size; i++)
//	{
//		h_data[i] = (int)(rand() & 0xff);
//	}
//	
//
//	for (int i = 0; i < array_size; i++)
//	{
//		printf("%d ", h_data[i]);
//		if (i % 16 == 15)
//			printf("\n");
//	}
//	printf("\n \n");
//
//	int* d_data;
//	cudaMalloc((void**)&d_data, array_byte_size);
//	cudaMemcpy(d_data, h_data, array_byte_size, cudaMemcpyHostToDevice);
//
//	dim3 block(2, 2, 2);
//	dim3 grid(2, 2, 2);
//
//	unique_gid_calculation_3d << <grid, block >> > (d_data);
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//	return 0;
//}