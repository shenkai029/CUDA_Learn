#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

#include "common.h"
#include "cuda_common.cuh"

__global__ void sum_array_gpu_3(int* a, int* b, int* c, int* res, size_t size)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size)
	{
		res[gid] = a[gid] + b[gid] + c[gid];
	}

}

void sum_array_cpu(int* a, int* b, int* c, int* res, size_t size)
{
	for (int i = 0; i < size; i++)
	{
		res[i] = a[i] + b[i] + c[i];
	}
}

//int main()
//{
//	int size = 10000000;
//	int block_size = 256;
//
//	int num_bytes = sizeof(int) * size;
//
//	// host pointers
//	int* h_a, * h_b, * h_c, * gpu_result, * cpu_result;
//
//	// allocate memory for host pointers
//	h_a = (int*)malloc(num_bytes);
//	h_b = (int*)malloc(num_bytes);
//	h_c = (int*)malloc(num_bytes);
//	gpu_result = (int*)malloc(num_bytes);
//	cpu_result = (int*)malloc(num_bytes);
//
//	// init host pointers
//	time_t t;
//	srand((unsigned)time(&t));
//	for (int i = 0; i < size; i++)
//	{
//		h_a[i] = (int)(rand() & 0x3fffff);
//	}
//
//	for (int i = 0; i < size; i++)
//	{
//		h_b[i] = (int)(rand() & 0x3fffff);
//	}
//
//	for (int i = 0; i < size; i++)
//	{
//		h_c[i] = (int)(rand() & 0x3fffff);
//	}
//
//	memset(gpu_result, 0, num_bytes);
//	memset(cpu_result, 0, num_bytes);
//
//	//summation in CPU
//	clock_t cpu_start, cpu_end;
//	cpu_start = clock();
//	sum_array_cpu(h_a, h_b, h_c, cpu_result, size);
//	cpu_end = clock();
//
//	// device pointers
//	int* d_a, * d_b, * d_c, * d_res;
//	gpuErrChk(hipMalloc((void**)&d_a, num_bytes));
//	gpuErrChk(hipMalloc((void**)&d_b, num_bytes));
//	gpuErrChk(hipMalloc((void**)&d_c, num_bytes));
//	gpuErrChk(hipMalloc((void**)&d_res, num_bytes));
//
//	//launching the grid
//	dim3 block(block_size);
//	dim3 grid((size / block.x) + 1);
//
//	clock_t htod_start, htod_end;
//	htod_start = clock();
//	hipMemcpy(d_a, h_a, num_bytes, hipMemcpyHostToDevice);
//	hipMemcpy(d_b, h_b, num_bytes, hipMemcpyHostToDevice);
//	hipMemcpy(d_c, h_c, num_bytes, hipMemcpyHostToDevice);
//	htod_end = clock();
//
//	clock_t gpu_start, gpu_end;
//	gpu_start = clock();
//	sum_array_gpu_3 << <grid, block >> > (d_a, d_b, d_c, d_res,size);
//	hipDeviceSynchronize();
//	gpu_end = clock();
//
//	// memory transfer back to host
//	clock_t dtoh_start, dtoh_end;
//	dtoh_start = clock();
//	hipMemcpy(gpu_result, d_res, num_bytes, hipMemcpyDeviceToHost);
//	dtoh_end = clock();
//
//	// array comparison
//	compare_arrays(gpu_result, cpu_result, size);
//
//	printf("Sum array CPU execution time: %2.8f \n",
//		(double)((double)(cpu_end - cpu_start) / CLOCKS_PER_SEC));
//
//	printf("Sum array GPU execution time: %2.8f \n",
//		(double)((double)(gpu_end - gpu_start) / CLOCKS_PER_SEC));
//
//	printf("htod mem transfer time: %2.8f \n",
//		(double)((double)(htod_end - htod_start) / CLOCKS_PER_SEC));
//
//	printf("dtoh mem transfer time: %2.8f \n",
//		(double)((double)(dtoh_end - dtoh_start) / CLOCKS_PER_SEC));
//
//	printf("Sum array GPU total execution time: %2.8f \n",
//		(double)((double)(dtoh_end - htod_start) / CLOCKS_PER_SEC));
//
//	hipFree(d_res);
//	hipFree(d_c);
//	hipFree(d_b);
//	hipFree(d_a);
//
//	free(cpu_result);
//	free(gpu_result);
//	free(h_c);
//	free(h_b);
//	free(h_a);
//
//	hipDeviceReset();
//	return 0;
//}